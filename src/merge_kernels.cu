#include "hip/hip_runtime.h"
#include "sm_pointcloud_merger_ros2/merge_kernels.cuh"
#include <hip/hip_runtime.h>
#include <sensor_msgs/msg/point_field.hpp>
#include <sensor_msgs/msg/point_cloud2.hpp>
#include <pcl_conversions/pcl_conversions.h>
#include <pcl/point_types.h>
#include <pcl/point_cloud.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

using sensor_msgs::msg::PointCloud2;

namespace
{
  // ────────────────────────────────────────────────────────────────────
  // CUDA 커널: 4×4 변환행렬을 적용하여 src → dst 복사
  // (행렬은 행 우선, 12원소 배열)
  // ────────────────────────────────────────────────────────────────────
  __global__ void transform_and_copy_kernel(
    const float4* __restrict__ src,
    float4* __restrict__ dst,
    const float* __restrict__ tf, int N)
  {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
      float4 p = src[i];
      float x = tf[0]*p.x + tf[1]*p.y + tf[2] *p.z + tf[3];
      float y = tf[4]*p.x + tf[5]*p.y + tf[6] *p.z + tf[7];
      float z = tf[8]*p.x + tf[9]*p.y + tf[10]*p.z + tf[11];
      dst[i] = make_float4(x, y, z, p.w);
    }
  }

  // 디바이스에 미리 올려둘 단순 4×4 단위 행렬
  __constant__ float IDENTITY_TF[12] = {1,0,0,0, 0,1,0,0, 0,0,1,0};

  // ────────────────────────────────────────────────────────────────────
  // 헬퍼: PCL→디바이스 float4 포인터 변환
  // ────────────────────────────────────────────────────────────────────
    struct DeviceCloud
    {
        thrust::device_vector<float4> data;
        int size() const { return static_cast<int>(data.size()); }
        float4* ptr()     { return thrust::raw_pointer_cast(data.data()); }
    };

    DeviceCloud to_device(const pcl::PointCloud<pcl::PointXYZI>& cloud)
    {
        // 1) 호스트에서 float4 배열 생성
        std::vector<float4> host_data;
        host_data.reserve(cloud.points.size());
        for (const auto &p : cloud.points) {
            float4 v;
            v.x = p.x; v.y = p.y; v.z = p.z; v.w = p.intensity;
            host_data.push_back(v);
        }

        // 2) 디바이스 벡터에 업로드
        DeviceCloud dev;
        dev.data = thrust::device_vector<float4>(
        host_data.begin(), host_data.end());
        return dev;
    }
}

// ──────────────────────────────────────────────────────────────────────
// 호스트 함수 구현
// ──────────────────────────────────────────────────────────────────────
void gpu_merge_pointclouds(
  const std::vector<PointCloud2::ConstSharedPtr>& input_clouds,
  PointCloud2& output)
{
  // 1. PointCloud2 → PCL 변환
  std::vector<pcl::PointCloud<pcl::PointXYZI>> pcl_vec(input_clouds.size());
  size_t total_points = 0;
  for (size_t i=0; i<input_clouds.size(); ++i)
  {
    pcl::PointCloud<pcl::PointXYZI> tmp;
    pcl::fromROSMsg(*input_clouds[i], tmp);
    total_points += tmp.size();
    pcl_vec[i] = std::move(tmp);
  }

  // 2. 디바이스 메모리 복사
  std::vector<DeviceCloud> dev_vec;
  dev_vec.reserve(pcl_vec.size());
  for (auto & pc : pcl_vec) dev_vec.emplace_back(to_device(pc));

  // 3. 결과 버퍼 생성
  thrust::device_vector<float4> merged_dev(total_points);

  // 4. 각 LiDAR 변환행렬(예: 단위행렬) 적용 및 큰 버퍼에 복사
  size_t offset = 0;
  const int threads = 256;
  for (auto & dc : dev_vec)
  {
    int blocks = (dc.size() + threads - 1) / threads;
    transform_and_copy_kernel<<<blocks, threads>>>(
      dc.ptr(),
      thrust::raw_pointer_cast(merged_dev.data()) + offset,
      IDENTITY_TF, dc.size());
    offset += dc.size();
  }
  hipDeviceSynchronize();

  // 5. 디바이스→호스트 float4 배열 복사
  std::vector<float4> host_data(total_points);
  thrust::copy(merged_dev.begin(), merged_dev.end(), host_data.begin());

  // 6. float4 → PCL PointXYZI 변환
  pcl::PointCloud<pcl::PointXYZI> merged_host;
  merged_host.points.resize(total_points);
  for (size_t i = 0; i < total_points; ++i) {
      const float4 &v = host_data[i];
      pcl::PointXYZI p;
      p.x = v.x; p.y = v.y; p.z = v.z; p.intensity = v.w;
      merged_host.points[i] = p;
  }
  // 7. PCL → PointCloud2 메시지로 변환 및 헤더 복사
  pcl::toROSMsg(merged_host, output);
  output.header = input_clouds.front()->header;
}
